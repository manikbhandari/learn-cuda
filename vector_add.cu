
#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const int DSIZE = 1 << 28;
const int numThreadsPerBlock = 256;  // CUDA maximum is 1024
// vector add kernel: C = A + B
__global__ void vadd(const float *A, const float *B, float *C, int ds){
  int idx = blockIdx.x * blockDim.x + threadIdx.x; // create typical 1D thread index from built-in variables
  if (idx < ds) {
        C[idx] = A[idx] + B[idx];
  }
}

int main(){
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[DSIZE];  // allocate space for vectors in host memory
    h_B = new float[DSIZE];
    h_C = new float[DSIZE];

    for (int i = 0; i < DSIZE; i++){  // initialize vectors in host memory
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
        h_C[i] = 0;
    }
    hipMalloc(&d_A, DSIZE * sizeof(float));  // allocate device space for vector A
    hipMalloc(&d_B, DSIZE * sizeof(float));  // allocate device space for vector B
    hipMalloc(&d_C, DSIZE * sizeof(float));  // allocate device space for vector C
    cudaCheckErrors("hipMalloc failure"); // error checking

    // copy vector A to device:
    hipMemcpy(d_A, h_A, DSIZE * sizeof(float), hipMemcpyHostToDevice);
    // copy vector B to device:
    hipMemcpy(d_B, h_B, DSIZE * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    // Take ceiling number of blocks => launch some extra threads
    const int numBlocks = (DSIZE + numThreadsPerBlock - 1) / numThreadsPerBlock;
    vadd<<<numBlocks, numThreadsPerBlock>>>(d_A, d_B, d_C, DSIZE);
    cudaCheckErrors("kernel launch failure");

    // copy vector C from device to host:
    hipMemcpy(h_C, d_C, DSIZE * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");

    // Why not synchronize here?
    printf("A[0] = %f\n", h_A[0]);
    printf("B[0] = %f\n", h_B[0]);
    printf("C[0] = %f\n", h_C[0]);
    return 0;
}
  