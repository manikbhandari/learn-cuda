/*
    Run the following command to get debug info:
    nvcc -g -G reduction.cu -o bin/reduction
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>

#define BLOCK_SZ 32
#define DEBUG false

using namespace std;

const float *getRandomMatrix(unsigned int X1)
{
    unsigned int numElements = X1;
    float *matrix = new float[numElements];
    for (unsigned int i = 0; i < numElements; i++)
    {
        matrix[i] = rand() / (float)RAND_MAX;
        if (DEBUG)
            matrix[i] = 1.0;
    }
    return (const float *)matrix;
}

void printMatrix(const float *matrix, unsigned int X1)
{
    if (matrix == nullptr)
    {
        cout << "Cannot print null matrix";
        return;
    }
    for (unsigned int i = 0; i < X1; i++)
    {
        cout << int(matrix[i]) << " ";
    }
}

float *cpu_reduction(const float *vec, unsigned int X1)
{
    float *ans = new float[1];
    for (int i = 0; i < X1; i++)
    {
        ans[0] += vec[i];
    }
    return ans;
}

__global__ void gpu_reduction(float *vec, unsigned int X1, float *output)
{
    __shared__ float vec_s[BLOCK_SZ];
    int t = threadIdx.x;
    int i = blockIdx.x * BLOCK_SZ + t;
    vec_s[t] = 0;
    if(i < X1)
        vec_s[t] = vec[i];
    __syncthreads();

    // TODO: this requires block_sz to be a power of 2
    for (int stride = BLOCK_SZ / 2; stride >= 1; stride /= 2)
    {
        if (t < stride)
            vec_s[t] = vec_s[t] + vec_s[t + stride];
        __syncthreads();
    }

    if (t == 0)
        atomicAdd(output, vec_s[0]);
}

int main()
{
    unsigned int X1 = DEBUG ? 32 : 512;
    printf("X1=%d \n", X1);
    const float *vec = getRandomMatrix(X1);

    if (DEBUG)
    {
        cout << "Matrix 1:" << endl;
        printMatrix(vec, X1);
        cout << endl;
    }
    clock_t t0 = clock();
    float *cpu_ans = cpu_reduction(vec, X1);
    clock_t t1 = clock();
    float cpu_time = ((double)(t1 - t0)) / CLOCKS_PER_SEC;
    printf("CPU stencil took %fs\n", cpu_time);
    if (DEBUG)
    {
        cout << "CPU Ans: " << *cpu_ans << endl;
    }

    clock_t t_gpu_overall_st = clock();
    // allocate memory on device
    float *vec_d;
    float *ans_d;
    hipMalloc(&vec_d, X1 * sizeof(float));
    hipMalloc(&ans_d, 1 * sizeof(float));
    hipDeviceSynchronize();

    // copy to device
    hipMemcpy(vec_d, vec, X1 * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // compute
    clock_t t2 = clock();
    dim3 numThreadsPerBlock(BLOCK_SZ); // 1024 is the max allowed value for this
    int nBlocks = (X1 + BLOCK_SZ - 1) / BLOCK_SZ;
    if (DEBUG)
    {
        printf("numBlocks=%d\n", nBlocks);
    }
    dim3 numBlocks(nBlocks);
    gpu_reduction<<<numBlocks, numThreadsPerBlock>>>(vec_d, X1, ans_d);
    hipDeviceSynchronize();

    // copy result to host
    float *gpu_ans = new float[1];
    gpu_ans[0] = 0;
    hipMemcpy(gpu_ans, ans_d, sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    clock_t t3 = clock();
    float gpu_time = ((double)(t3 - t2)) / CLOCKS_PER_SEC;
    printf("GPU stencil took %fs\n", gpu_time);

    // free memory on device
    hipFree(vec_d);
    hipFree(ans_d);
    hipDeviceSynchronize();
    clock_t t_gpu_overall_en = clock();
    float gpu_overall_time = ((double)(t_gpu_overall_en - t_gpu_overall_st)) / CLOCKS_PER_SEC;
    printf("Overall GPU stencil took %fs\n", gpu_overall_time);

    // Check correctness
    float eps = 1e-2;
    if (fabs(*cpu_ans - *gpu_ans) >= eps)
    {
        printf("cpu_ans has %f but gpu_ans has %f\n", *cpu_ans, *gpu_ans);
        return 0;
    }

    return 0;
}
